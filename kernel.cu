#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <iostream>
#include <ctime>
#define EPS 1e-5
#define BND 1e5
#define TEST_LINE_LEFT_ID -1
#define TEST_LINE_RIGHT_ID -2
#define TAG_I_PLUS 1
#define TAG_I_MINUS -1
#define TAG_I_ZERO 0

#include "input_output.h"
#include "PerformanceTimer.h"

struct Line
{
	double a, b, c, slope;
	int id, tag;

	__host__ __device__
		Line() { a = 0; b = 0; c = 0; slope = 0; id = -3; tag = -2; }

	__host__ __device__
		Line(double aa, double bb, double cc, int index=-3) {
		a = aa; 
		b = bb; 
		c = cc; 
		id = index;
		if (b > EPS) tag = 1;
		else if (b < -EPS) tag = -1;
		else tag = 0;
		slope = -a / b;
	}

	__host__ __device__
	Line(const Line& L) {
		a = L.a;
		b = L.b;
		c = L.c;
		id = L.id;
		tag = L.tag;
		slope = L.slope;
	}
	
	friend std::ostream &operator<<(std::ostream &stream, const Line &p) {
		stream << p.id << ":(" << p.a << "," << p.b << "," << p.c << ")";
		return stream;
	}
};

struct Point
{
	double x, y;
	int i, j;

	__host__ __device__
		Point() { x = 0; y = 0; i = -3; j = -3; }

	__host__ __device__
		Point(double xx, double yy, int ii, int jj) { x = xx; y = yy; i = ii; j = jj; }

	__host__ __device__
		Point(const Point& p) {
		x = p.x;
		y = p.y;
		i = p.i;
		j = p.j;
	}

	friend std::ostream &operator<<(std::ostream& stream, const Point& p) {
		stream << "[" << p.x << "," << p.y << "]<-(" << p.i << "," << p.j << ")";
		return stream;
	}
};

struct test_line_ip
{
	int cross_line_id, tag;
	double x, y, slope;

	__host__ __device__
		test_line_ip() { cross_line_id = -3; tag = -2; x = 0; y = 0; slope = 0; }

	__host__ __device__
		test_line_ip(int id, int ttag, double xx, double yy, double k) {
		cross_line_id = id;
		tag = ttag;
		x = xx;
		y = yy;
		slope = k;
	}

	__host__ __device__
		test_line_ip(const test_line_ip& p) {
		cross_line_id = p.cross_line_id;
		tag = p.tag;
		x = p.x;
		y = p.y;
		slope = p.slope;
	}

	friend std::ostream &operator<<(std::ostream& stream, const test_line_ip& p) {
		stream << "[" << p.x << "," << p.y << "]:" << p.cross_line_id  << ":" << p.tag << ":" << p.slope;
		return stream;
	}
};

struct compute_test_line_ip
{
	Line test;

	compute_test_line_ip(const Line& l) { test = l; }

	__host__ __device__
		test_line_ip operator() (const Line& line) {
		double a1, a2, b1, b2, c1, c2;
		a1 = test.a;
		b1 = test.b;
		c1 = test.c;
		a2 = line.a;
		b2 = line.b;
		c2 = line.c;
		double x = (c1 * b2 - b1 * c2) / (a1 * b2 - b1 * a2);
		double y = (c1 * a2 - a1 * c2) / (b1 * a2 - a1 * b2);
		return test_line_ip(line.id, line.tag, x, y, line.slope);
	}
};

struct max_iplus
{
	__host__ __device__
		test_line_ip operator() (const test_line_ip& pa, const test_line_ip& pb) {
		if (pa.tag == TAG_I_PLUS && pb.tag == TAG_I_PLUS) {
			return pa.y + EPS < pb.y ? pb : pa;
		}
		else if (pa.tag == TAG_I_PLUS && pb.tag == TAG_I_MINUS) {
			return pa;
		}
		else{
			return pb;
		}
	}
};

struct min_minus
{
	__host__ __device__
		test_line_ip operator() (const test_line_ip& pa, const test_line_ip& pb) {
		if (pa.tag == TAG_I_MINUS && pb.tag == TAG_I_MINUS) {
			return pa.y + EPS < pb.y ? pa : pb;
		}
		else if (pa.tag == TAG_I_MINUS && pb.tag == TAG_I_PLUS) {
			return pa;
		}
		else {
			return pb;
		}
	}
};

struct min_max_pair
{
	test_line_ip max_up;
	test_line_ip min_down;

	min_max_pair() {}

	min_max_pair(test_line_ip mu, test_line_ip md) {
		max_up = mu;
		min_down = md;
	}
};

struct rotate_line
{
	double A, B;

	rotate_line(double obj_a, double obj_b) { A = obj_a; B = obj_b; }

	__host__ __device__
		Line operator() (const Line& line) {
		double a = line.a, b = line.b;

		// rotate the normal vector of line
		double new_a, new_b;
		// new_a = (B,-A) * (a,b) / sqrt(A^2+B^2)
		new_a = (B*a - A*b) / sqrt(A*A + B*B);
		// new_b = (A,B) * (a,b) / sqrt(A^2+B^2)
		new_b = (A*a + B*b) / sqrt(A*A + B*B);
		// rotate the normal vector of line

		return Line(new_a, new_b, line.c, line.id);
	}
};

template<typename T>
void print(const thrust::device_vector<T>& vec) {
	thrust::copy(vec.begin(), vec.end(), std::ostream_iterator<T>(std::cout, "\n"));
	std::cout << std::endl;
}

template <typename T>
void print(const T& obj) {
	std::cout << obj << std::endl;
}

void println() {
	std::cout << std::endl;
}

Point compute_ip(const Line& line1, const Line& line2) {
	double a1, a2, b1, b2, c1, c2;
	a1 = line1.a;
	b1 = line1.b;
	c1 = line1.c;
	a2 = line2.a;
	b2 = line2.b;
	c2 = line2.c;
	double x = (c1 * b2 - b1 * c2) / (a1 * b2 - b1 * a2);
	double y = (c1 * a2 - a1 * c2) / (b1 * a2 - a1 * b2);
	return Point(x, y, line1.id, line2.id);
}

min_max_pair test(double test_x, const thrust::device_vector<Line>& lines, thrust::device_vector<test_line_ip>& test_ips) {
	// calculate all ips with the test line
	Line test_line(1, 0, test_x);
	thrust::transform(lines.begin(), lines.end(), test_ips.begin(), compute_test_line_ip(test_line));
	
	// find the highest I+ line and the lowest I- line
	test_line_ip init_ip = test_ips[0];
	test_line_ip p_up_max = thrust::reduce(test_ips.begin(), test_ips.end(), init_ip, max_iplus());
	test_line_ip p_down_min = thrust::reduce(test_ips.begin(), test_ips.end(), init_ip, min_minus());

	return min_max_pair(p_up_max, p_down_min);
}

min_max_pair find_boundary(double start, const thrust::device_vector<Line>& lines, thrust::device_vector<test_line_ip>& test_ips) {
	double bnd = start;
	min_max_pair mmp;
	Line line1, line2;
	Point ip;
	while (true)
	{
		mmp = test(bnd, lines, test_ips);
		// if there is I- lines and min I- is below the max I+
		if (mmp.min_down.tag == TAG_I_MINUS && mmp.min_down.y + EPS < mmp.max_up.y) {
			// move left boundary to their ip
			line1 = lines[mmp.min_down.cross_line_id];
			line2 = lines[mmp.max_up.cross_line_id];
			ip = compute_ip(line1, line2);
			bnd = ip.x;
		}
		else
		{
			return mmp;
		}
	}
}

void rotate(double* point, double A, double B) {
	double x, y;
	x = point[0];
	y = point[1];

	// rotate the normal vector of line
	double new_a, new_b;
	// new_a = (B,-A) * (a,b) / sqrt(A^2+B^2)
	new_a = (B*x - A*y) / sqrt(A*A + B*B);
	// new_b = (A,B) * (a,b) / sqrt(A^2+B^2)
	new_b = (A*x + B*y) / sqrt(A*A + B*B);

	point[0] = new_a;
	point[1] = new_b;
}

double ans_x, ans_y, left = -BND, right = BND;
test_line_ip left_max_up, right_max_up, tmp_max_up;
min_max_pair left_minmax, right_minmax, mmp;
Line left_line, right_line;
Point next_test_ip;


int main() {
	/* No line removing version */
	// read in all data
	inputs * input = read_from_file("./testcases/100000_0.dat");
	int ans_line_i, ans_line_j, N = input->number;

	// load data from cpu to gpu
	thrust::host_vector<Line> h_lines(N);
	for (int i = 0; i < N; i++) {
		h_lines[i] = Line(input->lines[i]->param_a, input->lines[i]->param_b, input->lines[i]->param_c, i);
	}
	thrust::device_vector<Line> lines = h_lines;
	thrust::device_vector<test_line_ip> test_ips(N);

	// start timeing
	print("Start working on GPU...");
	double start = get_cpu_time();

	// rotate lines
	thrust::transform(lines.begin(), lines.end(), lines.begin(), rotate_line(input->obj_function_param_a, input->obj_function_param_b));

	// first step: narrow down the left and right boundary until both reach the feasiable region
	left_minmax = find_boundary(left, lines, test_ips);
	right_minmax = find_boundary(right, lines, test_ips);
	left_max_up = left_minmax.max_up;
	right_max_up = right_minmax.max_up;

	// second stpe: make sure we do not miss the special cases where the optim point is an ip of a I+ and I-
	if (left_max_up.slope > EPS) {
		ans_x = left_max_up.x;
		ans_y = left_max_up.y;
		ans_line_i = left_max_up.cross_line_id;
		ans_line_j = left_minmax.min_down.cross_line_id;
	}
	else if (right_max_up.slope < -EPS)
	{
		ans_x = right_max_up.x;
		ans_y = right_max_up.y;
		ans_line_i = right_max_up.cross_line_id;
		ans_line_j = right_minmax.min_down.cross_line_id;
	}
	else
	{
		// third step: find new test line between two boundaries and update the L/R boundary until reach the new test exactly hit the optim
		while (true)
		{
			left_line = lines[left_max_up.cross_line_id];
			right_line = lines[right_max_up.cross_line_id];
			next_test_ip = compute_ip(left_line, right_line);
			mmp = test(next_test_ip.x, lines, test_ips);
			tmp_max_up = mmp.max_up;
			if (tmp_max_up.cross_line_id == left_line.id || tmp_max_up.cross_line_id == right_line.id) {
				ans_x = tmp_max_up.x;
				ans_y = tmp_max_up.y;
				ans_line_i = left_line.id;
				ans_line_j = right_line.id;
				print("find answer between the two boundaries.");
				break;
			}
			else
			{
				if (tmp_max_up.slope > EPS) {
					right_max_up = tmp_max_up;
				}
				else if (tmp_max_up.slope < -EPS)
				{
					left_max_up = tmp_max_up;
				}
				else
				{
					print("find answer between the two boundaries and on a horizontal line!");
					print(tmp_max_up.x);
					print(tmp_max_up.y);
					exit(0);
				}
			}
		}
	}

	double end = get_cpu_time();
	print("End working on GPU...");
	
	double A = input->obj_function_param_a;
	double B = input->obj_function_param_b;
	double reverse_A = -A / sqrt(A*A + B*B);
	double reverse_B = B / sqrt(A*A + B*B);

	double ip[2] = { ans_x, ans_y };
	rotate(ip, reverse_A, reverse_B);

	answer * ans = (answer *)malloc(sizeof(answer));
	ans->intersection_point = (point *)malloc(sizeof(point));
	ans->intersection_point->pos_x = ip[0];
	ans->intersection_point->pos_y = ip[1];
	ans->line1 = input->lines[ans_line_i];
	ans->line2 = input->lines[ans_line_j];
	ans->answer_b = A * ip[0] + B * ip[1];

	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	printf("Time Used:%lf\n", end - start);

	return 0;
}